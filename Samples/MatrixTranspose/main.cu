#include "kernels.cuh"
#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <helper_functions.h>

#include <iostream>

// Current Problems：
// 1. No completely free of bank conflicts

bool checkAnswer(float *matA, float *matB, const int &NX, const int &NY) {
  for (int j = 0; j < NY; ++j) {
    for (int i = 0; i < NX; ++i) {
      if (matA[i * NY + j] != matB[j * NX + i]) {
        return false;
      }
    }
  }
  return true;
}

void printMat(float *mat, const int &NX, const int &NY) {
  for (int j = 0; j < NY; ++j) {
    for (int i = 0; i < NX; ++i) {
      std::cout << mat[j * NX + i] << " ";
    }
    std::cout << "\n";
  }
  std::cout << "\n";
}

int main() {
  const int NX = 1024;
  const int NY = 1024;
  const int NTIMES = 1;

  const int BLOCKSIZE = 32;
  dim3 Grid((NX + BLOCKSIZE - 1) / BLOCKSIZE, (NY + BLOCKSIZE - 1) / BLOCKSIZE,
            1);
  dim3 Block(BLOCKSIZE, BLOCKSIZE, 1);

  float *matA_d;
  float *matB_d;
  checkCudaErrors(hipMalloc((void **)&matA_d, sizeof(float) * NX * NY));
  checkCudaErrors(hipMalloc((void **)&matB_d, sizeof(float) * NX * NY));

  float *matA_h = new float[NX * NY];
  float *matB_h = new float[NX * NY];
  float *ans_h = new float[NX * NY];

  for (int j = 0; j < NY; ++j) {
    for (int i = 0; i < NX; ++i) {
      matB_h[j * NX + i] = j * NX + i;
    }
  }

  checkCudaErrors(hipMemcpy(matB_d, matB_h, sizeof(float) * NX * NY,
                             hipMemcpyHostToDevice));

  // transpose on host
  for (int j = 0; j < NY; ++j) {
    for (int i = 0; i < NX; ++i) {
      matA_h[i * NY + j] = matB_h[j * NX + i];
    }
  }

  auto kernel = &matrixCopyShared<BLOCKSIZE>;
  auto mem_size = static_cast<size_t>(sizeof(float) * NX * NY);
  hipEvent_t start, end;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&end));

  for (int i = 0; i < 5; ++i) {
    if (i == 0) {
      printf("Testing Kernel matrixCopyShared......\n");
    } else if (i == 1) {
      printf("Testing Kernel matrixTransposeNaive......\n");
      kernel = &matrixTransposeNaive;
    } else if (i == 2) {
      printf("Testing Kernel matrixTransposeShared......\n");
      kernel = &matrixTransposeShared<BLOCKSIZE>;
    } else if (i == 3) {
      printf("Testing Kernel matrixTransposeTransposeSharedPadding......\n");
      kernel = &matrixTransposeSharedPadding<BLOCKSIZE>;
    } else {
      printf("Testing Kernel matrixTransposeSharedSwizz......\n");
      kernel = &matrixTransposeSharedSwizz<BLOCKSIZE>;
    }
    // warm up
    kernel<<<Grid, Block>>>(matA_d, matB_d, NX, NY);
    checkCudaErrors(hipGetLastError());

    // start time measurements
    checkCudaErrors(hipEventRecord(start));
    for (int i = 0; i < NTIMES; ++i) {
      kernel<<<Grid, Block>>>(matA_d, matB_d, NX, NY);
      checkCudaErrors(hipGetLastError());
    }

    checkCudaErrors(hipEventRecord(end));
    checkCudaErrors(hipEventSynchronize(end));

    float kernelTime;
    checkCudaErrors(hipEventElapsedTime(&kernelTime, start, end));

    float kernelBandwidth = 2.0f * 1000.0f * mem_size / (1024 * 1024 * 1024) /
                            (kernelTime / NTIMES);
    printf("Effective throughput = %.4f GB/s\n", kernelBandwidth);

    checkCudaErrors(hipMemcpy(ans_h, matA_d, sizeof(float) * NX * NY,
                               hipMemcpyDeviceToHost));

    if (checkAnswer(ans_h, matB_h, NX, NY)) {
      std::cout << "right answer" << "\n";
    } else {
      std::cout << "wrong answer" << "\n";
    }
  }

  checkCudaErrors(hipFree(matA_d));
  checkCudaErrors(hipFree(matB_d));
  free(matA_h);
  free(matB_h);
}