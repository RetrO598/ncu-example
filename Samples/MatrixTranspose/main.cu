#include "kernels.cuh"
#include <cstdio>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <helper_functions.h>

#include <iostream>

// Current Problems：
// 1. No completely free of bank conflicts

bool checkAnswer(float *matA, float *matB, const int &NX, const int &NY) {
  for (int j = 0; j < NY; ++j) {
    for (int i = 0; i < NX; ++i) {
      if (matA[i * NY + j] != matB[j * NX + i]) {
        return false;
      }
    }
  }
  return true;
}

void printMat(float *mat, const int &NX, const int &NY) {
  for (int j = 0; j < NY; ++j) {
    for (int i = 0; i < NX; ++i) {
      std::cout << mat[j * NX + i] << " ";
    }
    std::cout << "\n";
  }
  std::cout << "\n";
}

int main() {
  const int NX = 1024;
  const int NY = 1024;

  const int BLOCKSIZE = 32;
  dim3 Grid((NX + BLOCKSIZE - 1) / BLOCKSIZE, (NY + BLOCKSIZE - 1) / BLOCKSIZE,
            1);
  dim3 Block(BLOCKSIZE, BLOCKSIZE, 1);

  float *matA_d;
  float *matB_d;
  checkCudaErrors(hipMalloc((void **)&matA_d, sizeof(float) * NX * NY));
  checkCudaErrors(hipMalloc((void **)&matB_d, sizeof(float) * NX * NY));

  float *matA_h = new float[NX * NY];
  float *matB_h = new float[NX * NY];
  float *ans_h = new float[NX * NY];

  for (int j = 0; j < NY; ++j) {
    for (int i = 0; i < NX; ++i) {
      matB_h[j * NX + i] = j * NX + i;
    }
  }

  checkCudaErrors(hipMemcpy(matB_d, matB_h, sizeof(float) * NX * NY,
                             hipMemcpyHostToDevice));

  // transpose on host
  for (int j = 0; j < NY; ++j) {
    for (int i = 0; i < NX; ++i) {
      matA_h[i * NY + j] = matB_h[j * NX + i];
    }
  }

  matrixTranposeSharedSwizz<<<Grid, Block>>>(matA_d, matB_d, NX, NY);

  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipMemcpy(ans_h, matA_d, sizeof(float) * NX * NY,
                             hipMemcpyDeviceToHost));

  // printMat(matB_h, NX, NY);
  // printMat(ans_h, NY, NX);
  if (checkAnswer(ans_h, matB_h, NX, NY)) {
    std::cout << "right answer" << "\n";
  } else {
    std::cout << "wrong answer" << "\n";
  }

  checkCudaErrors(hipFree(matA_d));
  checkCudaErrors(hipFree(matB_d));
  free(matA_h);
  free(matB_h);
}